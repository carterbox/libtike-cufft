#include "hip/hip_runtime.h"
#include "ptychofft.cuh"
#include "kernels.cuh"

// constructor, memory allocation
ptychofft::ptychofft(size_t ptheta, size_t nz, size_t n, size_t nscan,
  size_t ndetx, size_t ndety, size_t nprb
) :
  ptheta(ptheta), nz(nz), n(n), nscan(nscan), ndetx(ndetx), ndety(ndety),
  nprb(nprb)
{
	// allocate memory on GPU
	hipMalloc((void **)&f, ptheta * nz * n * sizeof(float2));
	hipMalloc((void **)&g, ptheta * nscan * ndetx * ndety * sizeof(float2));
	hipMalloc((void **)&scanx, ptheta * nscan * sizeof(float));
	hipMalloc((void **)&scany, ptheta * nscan * sizeof(float));
	hipMalloc((void **)&shiftx, ptheta * nscan * sizeof(float2));
	hipMalloc((void **)&shifty, ptheta * nscan * sizeof(float2));
	hipMalloc((void **)&prb, ptheta * nprb * nprb * sizeof(float2));

	// create batched 2d FFT plan on GPU with sizes (ndetx,ndety)
	int ffts[2];
	ffts[0] = ndetx;
	ffts[1] = ndety;
	hipfftPlanMany(&plan2d, 2, ffts, ffts, 1, ndetx * ndety, ffts, 1, ndetx * ndety, HIPFFT_C2C, ptheta * nscan);

	// create batched 2d FFT plan on GPU with sizes (nprb,nprb)	acting on arrays with sizes (ndetx,ndety)
	ffts[0] = nprb;
	ffts[1] = nprb;
	int inembed[2];
	inembed[0] = ndetx;
	inembed[1] = ndety;
	hipfftPlanMany(&plan2dshift, 2, ffts, inembed, 1, ndetx * ndety, inembed, 1, ndetx * ndety, HIPFFT_C2C, ptheta * nscan);

	// init 3d thread block on GPU
	BS3d.x = 32;
	BS3d.y = 32;
	BS3d.z = 1;

	// init 3d thread grids	on GPU
	GS3d0.x = ceil(nprb * nprb / (float)BS3d.x);
	GS3d0.y = ceil(nscan / (float)BS3d.y);
	GS3d0.z = ceil(ptheta / (float)BS3d.z);

	GS3d1.x = ceil(ndetx * ndety / (float)BS3d.x);
	GS3d1.y = ceil(nscan / (float)BS3d.y);
	GS3d1.z = ceil(ptheta / (float)BS3d.z);

	GS3d2.x = ceil(nscan / (float)BS3d.x);
	GS3d2.y = ceil(ptheta / (float)BS3d.y);
	GS3d2.z = 1;
}

// destructor, memory deallocation
ptychofft::~ptychofft()
{
  free();
}

void ptychofft::free()
{
  if(!is_free)
  {
    hipFree(f);
    hipFree(g);
    hipFree(scanx);
    hipFree(scany);
    hipFree(shiftx);
    hipFree(shifty);
    hipFree(prb);
    hipfftDestroy(plan2d);
    is_free = true;
  }
}

// forward ptychography operator g = FQf
void ptychofft::fwd(size_t g_, size_t f_, size_t scan_, size_t prb_)
{
	// copy arrays to GPU
	hipMemcpy(f, (float2 *)f_, ptheta * nz * n * sizeof(float2), hipMemcpyDefault);
	hipMemset(g, 0, ptheta * nscan * ndetx * ndety * sizeof(float2));
	hipMemcpy(scanx, &((float *)scan_)[0], ptheta * nscan * sizeof(float), hipMemcpyDefault);
	hipMemcpy(scany, &((float *)scan_)[ptheta * nscan], ptheta * nscan * sizeof(float), hipMemcpyDefault);
	hipMemcpy(prb, (float2 *)prb_, ptheta * nprb * nprb * sizeof(float2), hipMemcpyDefault);

	// take part for the probe multiplication and shift it via FFT
	takepart<<<GS3d0, BS3d>>>(g, f, prb, scanx, scany, ptheta, nz, n, nscan, nprb, ndetx, ndety);

	//// SHIFT start
	// Fourier transform
	hipfftExecC2C(plan2dshift, (hipfftComplex *)g, (hipfftComplex *)g, HIPFFT_FORWARD);
	// compute exp(1j dx),exp(1j dy) where dx,dy are in (-1,1) and correspond to shifts to nearest integer
	takeshifts<<<GS3d2, BS3d>>>(shiftx, shifty, scanx, scany, 1, ptheta, nscan);
	// perform shifts in the frequency domain by multiplication with exp(1j dx),exp(1j dy)
	shifts<<<GS3d1, BS3d>>>(g, shiftx, shifty, ptheta, nscan, ndetx * ndety, nprb*nprb);
	// inverse Fourier transform
	hipfftExecC2C(plan2dshift, (hipfftComplex *)g, (hipfftComplex *)g, HIPFFT_BACKWARD);
	//// SHIFT end

	// probe multiplication of the object array
	mulprobe<<<GS3d0, BS3d>>>(g, f, prb, scanx, scany, ptheta, nz, n, nscan, nprb, ndetx, ndety);
	// Fourier transform
	hipfftExecC2C(plan2d, (hipfftComplex *)g, (hipfftComplex *)g, HIPFFT_FORWARD);

	// copy result to CPU
	hipMemcpy((float2 *)g_, g, ptheta * nscan * ndetx * ndety * sizeof(float2), hipMemcpyDefault);
}

// adjoint ptychography operator with respect to object (flg==0) f = Q*F*g, or probe (flg==1) prb = Q*F*g
void ptychofft::adj(size_t f_, size_t g_, size_t scan_, size_t prb_, int flg)
{
	// copy arrays to GPU
	hipMemcpy(f, (float2 *)f_, ptheta * nz * n * sizeof(float2),hipMemcpyDefault);
	hipMemcpy(g, (float2 *)g_, ptheta * nscan * ndetx * ndety * sizeof(float2), hipMemcpyDefault);
	hipMemcpy(scanx, &((float *)scan_)[0], ptheta * nscan * sizeof(float), hipMemcpyDefault);
	hipMemcpy(scany, &((float *)scan_)[ptheta * nscan], ptheta * nscan * sizeof(float), hipMemcpyDefault);
	hipMemcpy(prb, (float2 *)prb_, ptheta * nprb * nprb * sizeof(float2), hipMemcpyDefault);

	// inverse Fourier transform
	hipfftExecC2C(plan2d, (hipfftComplex *)g, (hipfftComplex *)g, HIPFFT_BACKWARD);
	if (flg == 0)// adjoint probe multiplication operator
	{
		mulaprobe<<<GS3d0, BS3d>>>(f, g, prb, scanx, scany, ptheta, nz, n, nscan, nprb, ndetx, ndety);

		//// SHIFT start
		// Fourier transform
		hipfftExecC2C(plan2dshift, (hipfftComplex *)g, (hipfftComplex *)g, HIPFFT_FORWARD);
		// compute exp(1j dx),exp(1j dy) where dx,dy are in (-1,1) and correspond to shifts to nearest integer
		takeshifts<<<GS3d2, BS3d>>>(shiftx, shifty, scanx, scany, -1, ptheta, nscan);
		// perform shifts in the frequency domain by multiplication with exp(-1j dx),exp(-1j dy) - backward
		shifts<<<GS3d1, BS3d>>>(g, shiftx, shifty, ptheta, nscan, ndetx * ndety, nprb*nprb);
		hipfftExecC2C(plan2dshift, (hipfftComplex *)g, (hipfftComplex *)g, HIPFFT_BACKWARD);
		//// SHIFT end

		setpartobj<<<GS3d0, BS3d>>>(f, g, prb, scanx, scany, ptheta, nz, n, nscan, nprb, ndetx, ndety);
		// copy result to CPU
		hipMemcpy((float2 *)f_, f, ptheta * nz * n * sizeof(float2), hipMemcpyDefault);
	}
	else if (flg == 1)// adjoint object multiplication operator
	{
		mulaobj<<<GS3d0, BS3d>>>(prb, g, f, scanx, scany, ptheta, nz, n, nscan, nprb, ndetx, ndety);

		//// SHIFT start
		// Fourier transform
		hipfftExecC2C(plan2dshift, (hipfftComplex *)g, (hipfftComplex *)g, HIPFFT_FORWARD);
		// compute exp(1j dx),exp(1j dy) where dx,dy are in (-1,1) and correspond to shifts to nearest integer
		takeshifts<<<GS3d2, BS3d>>>(shiftx, shifty, scanx, scany, -1, ptheta, nscan);
		// perform shifts in the frequency domain by multiplication with exp(-1j dx),exp(-1j dy) - backward
		shifts<<<GS3d1, BS3d>>>(g, shiftx, shifty, ptheta, nscan, ndetx * ndety, nprb*nprb);
		hipfftExecC2C(plan2dshift, (hipfftComplex *)g, (hipfftComplex *)g, HIPFFT_BACKWARD);
		//// SHIFT end

		setpartprobe<<<GS3d0, BS3d>>>(prb, g, f, scanx, scany, ptheta, nz, n, nscan, nprb, ndetx, ndety);
		// copy result to CPU
		hipMemcpy((float2 *)prb_, prb, ptheta * nprb * nprb * sizeof(float2), hipMemcpyDefault);
	}
}
